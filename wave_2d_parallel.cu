#include "hip/hip_runtime.h"
#define _XOPEN_SOURCE 600
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <iostream>

// TASK: T1
// Include the cooperative groups library
// BEGIN: T1
#include <hip/hip_cooperative_groups.h>
// END: T1


// Convert 'struct timeval' into seconds in double prec. floating point
#define WALLTIME(t) ((double)(t).tv_sec + 1e-6 * (double)(t).tv_usec)

// Option to change numerical precision
typedef int64_t int_t;
typedef double real_t;

// TASK: T1b
// Variables needed for implementation
// BEGIN: T1b

namespace cg = cooperative_groups;
hipDeviceProp_t device_prop;

// Device buffers for three time steps, indexed with 2 ghost points for the boundary
real_t *d_U_prv, *d_U_cur, *d_U_nxt;

#define d_U_prv(i,j) d_U_prev[0][((i)+1)*(N+2)+(j)+1]
#define d_U(i,j)     d_U_cur[1][((i)+1)*(N+2)+(j)+1]
#define d_U_nxt(i,j) d_U_nxt[2][((i)+1)*(N+2)+(j)+1]

// Simulation parameters: size, step count, and how often to save the state
int_t
    N = 128,
    M = 128,
    max_iteration = 1000000,
    snapshot_freq = 1000;

// Wave equation parameters, time step is derived from the space step
const real_t
    c  = 1.0,
    dx = 1.0,
    dy = 1.0;
real_t
    dt;

// Declare the grid dimensions as constants on the device
// Since we don't specify grid dimensions as arguments when running the program,
// we could just set them as constexpr variables on the host side, but I used __constant__ memory instead
// to use more CUDA features.
__constant__ int_t d_N;
__constant__ int_t d_M;

__constant__ real_t d_dt, d_c, d_dx, d_dy;

// We only need the current time step on the host side
real_t *h_U_cur;

#define h_U(i,j) h_U_cur[((i)+1)*(N+2)+(j)+1]
// END: T1b

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


// Rotate the time step buffers.
void move_buffer_window ( void )
{
    real_t *temp = buffers[0];
    buffers[0] = buffers[1];
    buffers[1] = buffers[2];
    buffers[2] = temp;
}


// Save the present time step in a numbered file under 'data/'
void domain_save ( int_t step )
{
    char filename[256];
    sprintf ( filename, "data/%.5ld.dat", step );
    FILE *out = fopen ( filename, "wb" );
    for ( int_t i=0; i<M; i++ )
    {
        fwrite ( &U(i,0), sizeof(real_t), N, out );
    }
    fclose ( out );
}


// TASK: T4
// Get rid of all the memory allocations
void domain_finalize ( void )
{
// BEGIN: T4
    // Free the host memory
    free(h_U_cur);

    // Free the device memory
    hipFree(d_U_prv);
    hipFree(d_U_cur);
    hipFree(d_U_nxt);
    hipFree(&d_N);
    hipFree(&d_M);
// END: T4
}

// __device__ function for handling boundary conditions
__device__ void apply_boundary_conditions(real_t *shared_U, int i, int j, int local_i, int local_j, int shared_block_size_x, int shared_block_size_y, int d_M, int d_N) {
    // Neumann boundary conditions for shared memory
    // Handle the boundaries only within valid thread ranges
    
    // Apply boundary conditions to the left and right edges
    if (local_j == 1 && j > 0) {
        shared_U[local_i * shared_block_size_y] = shared_U[local_i * shared_block_size_y + 1];
    }
    if (local_j == shared_block_size_y - 2 && j < d_N - 1) {
        shared_U[local_i * shared_block_size_y + (shared_block_size_y - 1)] = shared_U[local_i * shared_block_size_y + (shared_block_size_y - 2)];
    }
    
    // Apply boundary conditions to the top and bottom edges
    if (local_i == 1 && i > 0) {
        shared_U[0 * shared_block_size_y + local_j] = shared_U[1 * shared_block_size_y + local_j];
    }
    if (local_i == shared_block_size_x - 2 && i < d_M - 1) {
        shared_U[(shared_block_size_x - 1) * shared_block_size_y + local_j] = shared_U[(shared_block_size_x - 2) * shared_block_size_y + local_j];
    }
}

// TASK: T6
// Neumann (reflective) boundary condition
// BEGIN: T6
void boundary_condition ( void )
{
    for ( int_t i=0; i<M; i++ )
    {
        U(i,-1) = U(i,1);
        U(i,N)  = U(i,N-2);
    }
    for ( int_t j=0; j<N; j++ )
    {
        U(-1,j) = U(1,j);
        U(M,j)  = U(M-2,j);
    }
}
// END: T6


// TASK: T5
__global__ void time_step_kernel(real_t *d_U_prv, real_t *d_U_cur, real_t *d_U_nxt) {
    // Define a shared memory tile with extra space for halo (boundary) elements
    extern __shared__ real_t shared_U[];

    // Calculate the global indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate local indices in shared memory
    int local_i = threadIdx.x + 1;
    int local_j = threadIdx.y + 1;

    // Get the dimensions of the block
    int shared_block_size_x = blockDim.x + 2; // +2 for the halo regions
    int shared_block_size_y = blockDim.y + 2; // +2 for the halo regions

    // Load current cell into shared memory
    if (i < d_M && j < d_N) {
        shared_U[local_i * shared_block_size_y + local_j] = d_U_cur[i * (d_N + 2) + j];

        // Load the halo cells (boundary values)
        if (threadIdx.x == 0 && i > 0) {
            shared_U[(local_i - 1) * shared_block_size_y + local_j] = d_U_cur[(i - 1) * (d_N + 2) + j];
        }
        if (threadIdx.x == blockDim.x - 1 && i < d_M - 1) {
            shared_U[(local_i + 1) * shared_block_size_y + local_j] = d_U_cur[(i + 1) * (d_N + 2) + j];
        }
        if (threadIdx.y == 0 && j > 0) {
            shared_U[local_i * shared_block_size_y + (local_j - 1)] = d_U_cur[i * (d_N + 2) + (j - 1)];
        }
        if (threadIdx.y == blockDim.y - 1 && j < d_N - 1) {
            shared_U[local_i * shared_block_size_y + (local_j + 1)] = d_U_cur[i * (d_N + 2) + (j + 1)];
        }
    }

    // Synchronize to make sure all threads have loaded their data into shared memory
    __syncthreads();

    // Apply boundary conditions using the __device__ function
    apply_boundary_conditions(shared_U, i, j, local_i, local_j, shared_block_size_x, shared_block_size_y, d_M, d_N);

    // Synchronize again to ensure boundary conditions are applied before computation
    __syncthreads();

    // Perform the calculation if within bounds
    if (i < d_M && j < d_N) {
        d_U_nxt[i * (d_N + 2) + j] = -d_U_prv[i * (d_N + 2) + j] + 2.0 * shared_U[local_i * shared_block_size_y + local_j]
            + (d_dt * d_dt * d_c * d_c) / (d_dx * d_dy) * (
                shared_U[(local_i - 1) * shared_block_size_y + local_j] +
                shared_U[(local_i + 1) * shared_block_size_y + local_j] +
                shared_U[local_i * shared_block_size_y + (local_j - 1)] +
                shared_U[local_i * shared_block_size_y + (local_j + 1)] -
                4.0 * shared_U[local_i * shared_block_size_y + local_j]
            );
    }
}
// Integration formula
// BEGIN; T5
void time_step ( void )
{

    
    

    for ( int_t i=0; i<M; i++ )
    {
        for ( int_t j=0; j<N; j++ )
        {
            U_nxt(i,j) = -U_prv(i,j) + 2.0*U(i,j)
                     + (dt*dt*c*c)/(dx*dy) * (
                        U(i-1,j)+U(i+1,j)+U(i,j-1)+U(i,j+1)-4.0*U(i,j)
                     );
        }
    }
}
// END: T5


// TASK: T7
// Main time integration.
void simulate( void )
{
// BEGIN: T7
    // Go through each time step
    for ( int_t iteration=0; iteration<=max_iteration; iteration++ )
    {
        if ( (iteration % snapshot_freq)==0 )
        {
            domain_save ( iteration / snapshot_freq );
        }

        // Derive step t+1 from steps t and t-1
        boundary_condition();
        time_step();

        // Rotate the time step buffers
        move_buffer_window();
    }
// END: T7
}


// TASK: T8
// GPU occupancy
void occupancy( void )
{
// BEGIN: T8
    ;
// END: T8
}


// TASK: T2
// Make sure at least one CUDA-capable device exists
static bool init_cuda()
{
// BEGIN: T2
     // Check the number of CUDA-capable devices.
    int device_count = 0;
    hipError_t error = hipGetDeviceCount(&device_count);
    if (error != hipSuccess || device_count == 0) {
        std::cerr << "No CUDA-compatible device found or failed to get device count: "
                  << hipGetErrorString(error) << std::endl;
        return false;
    }

    std::cout << "Number of CUDA-compatible devices: " << device_count << std::endl;

    // Iterate through devices and select a suitable one.
    for (int device = 0; device < device_count; ++device) {
        error = hipGetDeviceProperties(&device_prop, device);
        if (error != hipSuccess) {
            std::cerr << "Failed to get properties for device " << device << ": "
                      << hipGetErrorString(error) << std::endl;
            return false;
        }

        // Print information about the device (similar to Figure 3).
        std::cout << "Device " << device << ": " << device_prop.name << std::endl;
        std::cout << "  Compute capability: " << device_prop.major << "." << device_prop.minor << std::endl;
        std::cout << "  Total global memory: " << device_prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
        std::cout << "  Shared memory per block: " << device_prop.sharedMemPerBlock / 1024 << " KB" << std::endl;
        std::cout << "  Registers per block: " << device_prop.regsPerBlock << std::endl;
        std::cout << "  Warp size: " << device_prop.warpSize << std::endl;
        std::cout << "  Max threads per block: " << device_prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Max threads dimensions: [" << device_prop.maxThreadsDim[0] << ", "
                  << device_prop.maxThreadsDim[1] << ", " << device_prop.maxThreadsDim[2] << "]" << std::endl;
        std::cout << "  Max grid size: [" << device_prop.maxGridSize[0] << ", "
                  << device_prop.maxGridSize[1] << ", " << device_prop.maxGridSize[2] << "]" << std::endl;

        // Select the device (you can customize which one to select).
        hipSetDevice(device);
    }
    return true;
// END: T2
}


// TASK: T3
// Set up our three buffers, and fill two with an initial perturbation
void domain_initialize ( void )
{
// BEGIN: T3
    bool locate_cuda = init_cuda();
    if (!locate_cuda)
    {
        exit( EXIT_FAILURE );
    }

    // We only need the current time step on the host
    h_U_cur = (real_t *) malloc ( (M+2)*(N+2)*sizeof(real_t) );

    for ( int_t i=0; i<M; i++ )
    {
        for ( int_t j=0; j<N; j++ )
        {
            // Calculate delta (radial distance) adjusted for M x N grid
            real_t delta = sqrt ( ((i - M/2.0) * (i - M/2.0)) / (real_t)M +
                                ((j - N/2.0) * (j - N/2.0)) / (real_t)N );
            h_U(i,j) = exp ( -4.0*delta*delta );
        }
    }

    // Set the time step for 2D case
    dt = dx*dy / (c * sqrt (dx*dx+dy*dy));

    // Allocate device memory for the three time steps
    cudaErrorCheck(hipMalloc((void **)&d_U_prv, (M+2)*(N+2)*sizeof(real_t)));
    cudaErrorCheck(hipMalloc((void **)&d_U_cur, (M+2)*(N+2)*sizeof(real_t)));
    cudaErrorCheck(hipMalloc((void **)&d_U_nxt, (M+2)*(N+2)*sizeof(real_t)));

    // Copy the initial conditions from host to device for d_U_prv and d_U_cur
    cudaErrorCheck(hipMemcpy(d_U_prv, h_U_cur, (M+2)*(N+2)*sizeof(real_t), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMemcpy(d_U_cur, h_U_cur, (M+2)*(N+2)*sizeof(real_t), hipMemcpyHostToDevice));

    cudaErrorCheck(hipMalloc((void **)&d_N, sizeof(int_t)));
    cudaErrorCheck(hipMalloc((void **)&d_M, sizeof(int_t)));

    // Copy the grid size constants to the device constant memory using hipMemcpyToSymbol
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_N), &N, sizeof(int_t)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_M), &M, sizeof(int_t)));

    // Copy the wave equation parameters to the device constant memory using hipMemcpyToSymbol
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(real_t)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_c), &c, sizeof(real_t)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_dx), &dx, sizeof(real_t)));
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_dy), &dy, sizeof(real_t)));

// END: T3
}


int main ( void )
{
    // Set up the initial state of the domain
    domain_initialize();

    struct timeval t_start, t_end;

    gettimeofday ( &t_start, NULL );
    simulate();
    gettimeofday ( &t_end, NULL );

    printf ( "Total elapsed time: %lf seconds\n",
        WALLTIME(t_end) - WALLTIME(t_start)
    );

    occupancy();

    // Clean up and shut down
    domain_finalize();
    exit ( EXIT_SUCCESS );
}
